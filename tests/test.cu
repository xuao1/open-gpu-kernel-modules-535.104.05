#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

int main(int argc, char *argv[]) {
    // if (argc != 2) {
    //     fprintf(stderr, "Usage: %s <memory size in MiB>\n", argv[0]);
    //     return -1;
    // }

    // Parse memory size from command line
    // size_t bytes = (size_t)atoll(argv[1]) * 1024 * 1024;
    size_t bytes = 100 * 1024 * 1024;
    char *d_memory;

    printf("Trying to allocate and memset %zu bytes (%zu MiB) of GPU memory\n", bytes, bytes / (1024 * 1024));

    printf("Allocating %zu bytes (%zu MiB) of GPU memory\n", bytes, bytes / (1024 * 1024));
    hipError_t status = hipMalloc((void**)&d_memory, bytes);    
    printf("Allocated %zu bytes (%zu MiB) of GPU memory\n", bytes, bytes / (1024 * 1024));

    if (status != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(status));
        return -1;
    } 
    else printf("Successfully allocated %zu bytes (%zu MiB) of GPU memory\n", bytes, bytes / (1024 * 1024));

    status = hipMemset(d_memory, 0, bytes);

    if (status != hipSuccess) {
        fprintf(stderr, "hipMemset failed: %s\n", hipGetErrorString(status));
        hipFree(d_memory);
        return -1;
    }
    else printf("Successfully memset %zu bytes (%zu MiB) of GPU memory\n", bytes, bytes / (1024 * 1024));

    // printf("Successfully allocated and memset %zu bytes (%zu MiB) of GPU memory\n", bytes, bytes / (1024 * 1024));
    
    status = hipFree(d_memory);

    if(status != hipSuccess){
        fprintf(stderr, "hipFree failed: %s\n", hipGetErrorString(status));
        return -1;
    }
    else printf("Successfully freed %zu bytes (%zu MiB) of GPU memory\n", bytes, bytes / (1024 * 1024));

    printf("Successfully allocated and memset %zu bytes (%zu MiB) of GPU memory\n", bytes, bytes / (1024 * 1024));

    return 0;
}
