#include <stdio.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <iostream>

__global__ void testKernel(int *a) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    a[idx] = a[idx] + 1;
}

int main() {
    int *d_a;
    int size = 256 * sizeof(int);
    int *a = (int*)malloc(size);

    for (int i = 0; i < 256; i++) {
        a[i] = i;
    }

    hipMalloc((void **)&d_a, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    std::cout << "60s..." << std::endl;
    sleep(60);

    testKernel<<<1, 256>>>(d_a);

    std::cout << "60s..." << std::endl;
    sleep(60);

    hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        printf("%d ", a[i]);
    }
    printf("\n");

    hipFree(d_a);
    free(a);
    return 0;
}
