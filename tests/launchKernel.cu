#include <hip/hip_runtime.h>
#include <iostream>
#include <unistd.h> 

__global__ void simpleKernel(int *data) {
    int idx = threadIdx.x;
    data[idx] = idx;
}

int main() {
    int *d_data;
    hipMalloc(&d_data, sizeof(int) * 256);

    void *kernelArgs[] = { &d_data };
    dim3 grid(1);
    dim3 block(256);

    std::cout << "60 s..." << std::endl;
    sleep(60);

    hipLaunchKernel(reinterpret_cast<const void*>((void*)simpleKernel), grid, block, kernelArgs, 0, NULL);

    std::cout << "60 s..." << std::endl;
    sleep(60);

    hipFree(d_data);

    return 0;
}
