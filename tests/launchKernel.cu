#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <iostream>

__global__ void testKernel(int *a) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    a[idx] = a[idx] + 1;
}

int main() {
    int *d_a;
    int size = 256 * sizeof(int);
    int *a = (int*)malloc(size);

    for (int i = 0; i < 256; i++) {
        a[i] = i;
    }

    hipMalloc((void **)&d_a, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    // std::cin.get();
    std::cout << "60s..." << std::endl;
    sleep(60);

    hipDeviceSynchronize();
    // testKernel<<<1, 256>>>(d_a);
    hipModuleLaunchKernel((void*)testKernel, dim3(1, 1, 1), dim3(256, 1, 1), (void**)&d_a, 0, NULL);
    hipDeviceSynchronize();

    // std::cin.get();
    std::cout << "60s..." << std::endl;
    sleep(60);

    hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        printf("%d ", a[i]);
    }
    printf("\n");

    hipFree(d_a);
    free(a);
    return 0;
}
