#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void testKernel(int *a) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    a[idx] = a[idx] + 1;
}

int main() {
    int *d_a;
    int size = 256 * sizeof(int);
    int *a = (int*)malloc(size);

    for (int i = 0; i < 256; i++) {
        a[i] = i;
    }

    hipMalloc((void **)&d_a, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    hipLaunchKernel(reinterpret_cast<const void*>((void*)&testKernel), dim3(1), dim3(256), (void**)&d_a, 0, NULL);

    hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        printf("%d ", a[i]);
    }
    printf("\n");

    hipFree(d_a);
    free(a);
    return 0;
}
