#include <hip/hip_runtime.h>
#include <iostream>

__global__ void simpleKernel(int *data) {
    int idx = threadIdx.x;
    data[idx] = idx;
}

int main() {
    int *d_data;
    hipMalloc(&d_data, sizeof(int) * 256);

    void *kernelArgs[] = { &d_data };
    dim3 grid(1);
    dim3 block(256);

    std::cout << "Before launch kernel..." << std::endl;
    std::cin.get();

    hipLaunchKernel(reinterpret_cast<const void*>((void*)simpleKernel), grid, block, kernelArgs, 0, NULL);

    std::cout << "After launcher kernel..." << std::endl;
    std::cin.get();

    hipFree(d_data);

    return 0;
}

